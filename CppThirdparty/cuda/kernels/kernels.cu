#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <iostream>
#include <ctime> //时间库，计时用

// 在CPU上运行的函数
void CPUFunction()
{
  printf("This function is defined to run on the CPU.\n");
}

// 在GPU上运行的函数
// __global__ 关键字表明该函数将在GPU上运行并可全局调用（既可以由CPU，也可以由GPU调用）
__global__ void GPUFunction()
{
  printf("This function is defined to run on the GPU.\n");
}

void printDeviceInfo();

// 核函数
__global__ void kernelIndex()
{
  // 当执行到第255个线程块的第1023个线程时，才输出
  if (threadIdx.x == 1023 && blockIdx.x == 255)
  {
    printf("GPU for!\n");                     // 输出 Success！
    printf("threadIdx.x: %d\n", threadIdx.x); // 输出线程ID
    printf("blockIdx.x: %d\n", blockIdx.x);   // 输出线程块ID
    // 沉睡00.1ms
    clock_t start = clock();
    while (clock() - start < CLOCKS_PER_SEC / 100000)
      ;
  }
}

// 相同功能的CPU函数，说明并行的好处
void kernelIndexCPU()
{
  for (int i = 0; i < 256; i++)
  {
    for (int j = 0; j < 1024; j++)
    {
      // 当执行到第255个线程块的第1023个线程时，才输出
      if (j == 1023 && i == 255)
      {
        printf("CPU for!\n");           // 输出 Success！
        printf("threadIdx.x: %d\n", j); // 输出线程ID
        printf("blockIdx.x: %d\n", i);  // 输出线程块ID
      }
      // 沉睡00.1ms
      clock_t start = clock();
      while (clock() - start < CLOCKS_PER_SEC / 100000)
        ;
    }
  }
}

__global__ void loop()
{
  // 在Grid中遍历所有thread
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  printf("%d ", i);
}

// 处理工作量小于线程数的情况
__global__ void some_kernel(int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < N) // 保证线程ID小于元素数量N
  {
    printf("%d ", idx + 100);
  }
}

// CUDA 核函数，所有元素乘2
__global__ void doubleElements(int *a, int N)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N)
  {
    a[i] *= 2;
  }
}

// 检查数组内所有元素的值是否均为偶数
bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i * 2)
      return false;
  }
  return true;
}

// 使用grid-stride循环，这样每个线程可以处理数组中的多个元素。
__global__ void doubleElements2(int *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x; // grid 的一个跨步,跳到下一个网格计算

  for (int i = idx; i < N; i += stride)
  {
    a[i] *= 2;
  }
}

// GPU 矩阵乘法
__global__ void matrixMulGPU(int *a, int *b, int *c, int N)
{
  int val = 0;

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < N && col < N)
  {
    for (int k = 0; k < N; ++k)
      val += a[row * N + k] * b[k * N + col];
    c[row * N + col] = val;
  }
}

// CPU矩阵乘法
void matrixMulCPU(int *a, int *b, int *c, int N)
{
  int val = 0;

  for (int row = 0; row < N; ++row)
    for (int col = 0; col < N; ++col)
    {
      val = 0;
      for (int k = 0; k < N; ++k)
        val += a[row * N + k] * b[k * N + col];
      c[row * N + col] = val;
    }
}

void dim3grid()
{
  int N = 64;
  int *a, *b, *c_cpu, *c_gpu;
  int size = N * N * sizeof(int); // Number of bytes of an N x N matrix
  // 分配内存
  // 数据
  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  // 结果
  hipMallocManaged(&c_cpu, size);
  hipMallocManaged(&c_gpu, size);

  // 初始化数组（用一维数组来表示矩阵）
  for (int row = 0; row < N; ++row)
    for (int col = 0; col < N; ++col)
    {
      a[row * N + col] = row;
      b[row * N + col] = col + 2;
      c_cpu[row * N + col] = 0;
      c_gpu[row * N + col] = 0;
    }

  dim3 threads_per_block(16, 16, 1); // 一个 16 * 16 的线程阵
  dim3 number_of_blocks((N / threads_per_block.x) + 1, (N / threads_per_block.y) + 1, 1);
  matrixMulGPU<<<number_of_blocks, threads_per_block>>>(a, b, c_gpu, N); // 执行核函数 GPU矩阵乘法
  hipDeviceSynchronize();
  matrixMulCPU(a, b, c_cpu, N); // 执行 CPU 版本的矩阵乘法

  // 比较 CPU 和 GPU 两种方法的计算结果是否一致
  bool error = false;
  for (int row = 0; row < N && !error; ++row)
    for (int col = 0; col < N && !error; ++col)
      if (c_cpu[row * N + col] != c_gpu[row * N + col])
      {
        printf("FOUND ERROR at c[%d][%d]\n", row, col);
        error = true;
        break;
      }
  if (!error)
    printf("Success!\n");

  // 释放内存
  hipFree(a);
  hipFree(b);
  hipFree(c_cpu);
  hipFree(c_gpu);
}

void CppCUDAUsage()
{
  CPUFunction(); // 调用CPU函数

  /* 1 调用GPU函数,通常把要运行在GPU上的函数称为kernel（核）函数
   * 启动核(kernel)函数时，我们必须使用 <<< ... >>> 语法向核函数传递两个必要的参数
   * CUDA线程的层次结构分为三层：Thread（线程）、Block（块）、Grid（网格），网格由块组成，块由线程组成
   * 使用 <<<...>>> 中第一个1表示1个线程块，第二个1表示每个线程块1个线程。
   */
  // 通过这个函数中的内容获取设备信息
  printDeviceInfo();
  GPUFunction<<<2, 2>>>(); // 在GPU中为核函数分配5个具有5个线程的线程块，将运行2*2=4次,且四次同时运算

  // 2 我们将在 CPU 上执行的代码称为 Host （主机）代码，而将在 GPU 上运行的代码称为 Device （设备）代码
  // 与其他并行化的代码类似，核函数启动方式为异步，即CPU代码将继续执行而不会等待核函数执行完成
  // 调用CUDA提供的函数hipDeviceSynchronize可以让Host代码(CPU)等待Device代码(GPU)执行完毕，再在CPU上继续执行
  hipDeviceSynchronize();

  // 3 CUDA 核函数在由一个或多个线程块组成的网格中执行，
  // 且每个线程块中均包含相同数量的一个或多个线程（每个线程块中的线程数量相同）
  // 在核函数中，可以通过两个变量来获取到索引： threadIdx.x （线程索引）和 blockIdx.x（线程块索引）
  clock_t start = clock(); // 获得当前时间
  kernelIndex<<<480, 1024>>>();
  hipDeviceSynchronize();
  clock_t timeGPU = clock();
  double pt = ((double)(timeGPU - start)) / CLOCKS_PER_SEC; // 当前时间-开始标记时间，转化为秒
  std::cout << "GPU run times " << pt << "s" << std::endl;

  // 4 用CUDA加速For循环，同样功能的代码看出来，GPU运算速度差距
  kernelIndexCPU();
  clock_t timeCPU = clock();
  pt = ((double)(timeCPU - timeGPU)) / CLOCKS_PER_SEC;
  std::cout << "CPU run times " << pt << "s" << std::endl;

  // 5 核函数中记录了每个块中线程数的变量是blockDim.x（一个线程块中包含的线程数量，每个块中包含的线程数都是一样的）。
  // 通过将此变量与blockIdx.x和threadIdx.x变量结合使用，并借助表达式threadIdx.x+blockIdx.x*blockDim.x计算线程ID
  loop<<<2, 5>>>();
  hipDeviceSynchronize();
  printf("\n");

  // 6 鉴于 GPU 的硬件特性，线程块中的线程数最好配置为 32 的倍数，但是在实际工作中，
  // 不可能每次配置参数的时候都手动去算一遍最佳配置，更何况并不是所有的数都是 32 的倍数，通过以下三个步骤轻松地解决：
  //   （1）配置参数，使线程总数超过实际工作所需的线程数
  //   （2）然后，在向核函数传递参数时传递一个用于表示要处理的数据集总大小或完成工作所需的总线程数 N
  //   （3）最后，计算网格内的线程索引后（使用 threadIdx + blockIdx*blockDim），判断该索引是否超过 N，
  // 只在不超过的情况下执行与核函数相关的工作。
  // 假设N是已知的
  // int N = 100000;
  int N = 10;
  // 把每个block中的thread数设为256
  size_t threads_per_block = 256;
  // 根据N和thread数量配置Block数量
  size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;
  some_kernel<<<number_of_blocks, threads_per_block>>>(N);
  hipDeviceSynchronize();
  printf("\n");

  // 7 分配可同时被GPU和CPU访问的内存
  // CUDA可以便捷地分配和释放既可用于 Host 也可被 Device 访问的内存。
  // 在 Host（CPU）中，我们一般适用malloc 和 free 来分配和释放内存，
  // 但这样分配的内存无法直接被Device（GPU）访问，所以在这里我们用hipMallocManaged 和 hipFree
  // 两个函数来分配和释放同时可被 Host 和 Device 访问的内存。
  N = 1000;
  int *a;
  size_t size = N * sizeof(int);
  // hipMallocManaged在统一内存中创建了一个托管内存池（CPU上有，GPU上也有），
  // 内存池中已分配的空间可以通过相同的指针直接被CPU和GPU访问，底层系统在统一的内存空间中自动地在设备和主机间进行传输。
  hipMallocManaged(&a, size); // 为a分配CPU和GPU空间
  for (int i = 0; i < N; ++i)
    a[i] = i;
  number_of_blocks = (N + threads_per_block - 1) / threads_per_block; // block的数量
  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);      // 执行核函数，GPU访问
  hipDeviceSynchronize();
  bool areDoubled = checkElementsAreDoubled(a, N); // 检查元素是否为复数，CPU访问
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");
  hipFree(a); // 释放由hipMallocManaged

  // 8 反之，如果一个网格中的线程数量可能会小于实际工作量的大小，在核函数中使用跨网格循环
  // CUDA 提供一个记录了网格中线程块数的变量：gridDim.x。然后可以利用它来计算网格中的总线程数，
  // 即网格中的线程块数乘以每个线程块中的线程数：gridDim.x * blockDim.x
  N = 100000;
  int *b;
  hipMallocManaged(&b, N * sizeof(int));
  for (int i = 0; i < N; ++i)
    b[i] = i;
  // 与上面的那个函数不同，32*256<N, 在一个线程里面循环，算不到的就跳到下一个网格计算
  doubleElements2<<<32, 256>>>(b, N);
  hipDeviceSynchronize();
  // TRUE说明每个元素都有运行到
  printf("All elements were doubled? %s\n", checkElementsAreDoubled(b, N) ? "TRUE" : "FALSE");

  // 9 CUDA 函数发生错误时会返回一个类型为 hipError_t 的变量，该变量可用于检查调用函数时是否发生错误。
  hipError_t syncErr, asyncErr; // 定义错误处理变量
  // 单块线程数大于1024（前面说过每个block的线程数不能超过1024）
  doubleElements2<<<32, 2048>>>(b, N); // 执行核函数
  syncErr = hipGetLastError();        // hipGetLastError函数可以用于捕获核函数执行期间发生的错误
  // 捕获同步期间发生的错误，检查后续同步 CPU 与 GPU 时 API 调用所返回的状态（例如 hipDeviceSynchronize）
  // 如果之前执行的某一个核函数失败了，则将会发生错误
  asyncErr = hipDeviceSynchronize();
  // 输出错误 说明：两个错误需分别设置（即每次运行时只保留一个错误）
  if (syncErr != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(syncErr));
  if (asyncErr != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(asyncErr));
  // 输出Error: invalid configuration argument
  hipFree(b);

  // 10 网格和线程块最多可以定义有 3 个维度，使用多个维度定义网格和线程块在处理具有多个维度的数据时可能很有效，例如二维矩阵。
  // 如果要定义二维或三维的网格或线程块，可以使用 CUDA 的 dim3 关键字来定义多维网格或块
  dim3grid();

  return;
}

void printDeviceInfo()
{
  int device_id = 0; // 选择第一个设备
  hipDeviceProp_t device_props;
  hipGetDeviceProperties(&device_props, device_id);

  // 查询设备的线程块和线程的维度限制
  int max_threads_per_block = device_props.maxThreadsPerBlock;
  int max_threads_per_multiprocessor = device_props.maxThreadsPerMultiProcessor;
  int max_blocks_per_multiprocessor = device_props.maxBlocksPerMultiProcessor;
  dim3 max_threads_dim = {device_props.maxThreadsDim[0], device_props.maxThreadsDim[1], device_props.maxThreadsDim[2]};
  dim3 max_grid_size = {device_props.maxGridSize[0], device_props.maxGridSize[1], device_props.maxGridSize[2]};
  printf("Device %d properties:\n", device_id);
  printf("  Max threads per block: %d\n", max_threads_per_block);
  printf("  Max threads per multiprocessor: %d\n", max_threads_per_multiprocessor);
  printf("  Max blocks per multiprocessor: %d\n", max_blocks_per_multiprocessor);
  printf("  Max thread dimensions: (%d, %d, %d)\n", max_threads_dim.x, max_threads_dim.y, max_threads_dim.z);
  printf("  Max grid size: (%d, %d, %d)\n", max_grid_size.x, max_grid_size.y, max_grid_size.z);
  // Device 0 properties:
  //   Max threads per block: 1024
  //   Max threads per multiprocessor: 1536
  //   Max blocks per multiprocessor: 16
  //   Max thread dimensions: (1024, 1024, 64)
  //   Max grid size: (2147483647, 65535, 65535)
  // 通过上述信息可以看到当前设备（3060）一个线程块最大能够使用的线程数量为1024,超出这个限制将不执行

  int num_sm = device_props.multiProcessorCount;                   // 获取SM数量
  int max_blocks_per_sm = device_props.maxBlocksPerMultiProcessor; // 每个SM上的最大线程块数
  int max_blocks = num_sm * max_blocks_per_sm;                     // 可以使用的最大线程块数量
  printf("Device %d properties:\n", device_id);
  printf("  Number of SMs: %d\n", num_sm);
  printf("  Max blocks per SM: %d\n", max_blocks_per_sm);
  printf("  Max blocks: %d\n", max_blocks);
  // Device 0 properties:
  //   Number of SMs: 30
  //   Max blocks per SM: 16
  //   Max blocks: 480
  // 看出你的GPU具有30个SM，每个SM上最多可以运行16个线程块，因此可以使用的最大线程块数量为480
  // 但是实际上用的时候我超出这个限制也没报错
}
